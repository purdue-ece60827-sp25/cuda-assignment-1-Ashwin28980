#include "hip/hip_runtime.h"

#include "cudaLib.cuh"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	//	Insert GPU SAXPY kernel code here
	i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < size)	y[i] += scale * x[i];
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	//	Insert code here
	// Initialize host and device varibales
	float * x, * y, * y_dup;	// host (cpu) variables
	float * x_d, * y_d;			// device (gpu) variables

	// Memory allocation for host variables
	x = (float *) malloc(vectorSize * sizeof(float));
	y = (float *) malloc(vectorSize * sizeof(float));
	y_dup = (float *) malloc(vectorSize * sizeof(float));

	if (x == NULL || y == NULL || y_dup == NULL) {
		printf("Unable to malloc memory ... Exiting!");
		return -1;
	}

	vectorInit(x, vectorSize);
	vectorInit(y, vectorSize);
	//	y_dup = y
	std::memcpy(y_dup, y, vectorSize * sizeof(float));
	float scale = 2.0f;

	// Allocate memory for device variables and copy values from host variables
	hipMalloc((void **) &x_d, vectorSize);
	hipMemcpy(x_d, x, vectorSize, hipMemcpyHostToDevice);
	hipMalloc((void **) &y_d, vectorSize);
	hipMemcpy(y_d, y, vectorSize, hipMemcpyHostToDevice);

	#ifndef DEBUG_PRINT_DISABLE 
		printf("\n Adding vectors : \n");
		printf(" scale = %f\n", scale);
		printf(" x = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", x[i]);
		}
		printf(" ... }\n");
		printf(" y = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", y[i]);
		}
		printf(" ... }\n");
	#endif

	// Run device code
	saxpy_gpu<<<ceil(vectorSize/256.0), 256>>>(x_d, y_d, scale, vectorSize);
	hipMemcpy(y, y_d, vectorSize, hipMemcpyDeviceToHost);

	#ifndef DEBUG_PRINT_DISABLE 
		printf(" y = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", y[i]);
		}
		printf(" ... }\n");
	#endif

	int errorCount = verifyVector(x, y, y_dup, scale, vectorSize);
	std::cout << "Found " << errorCount << " / " << vectorSize << " errors \n";

	hipFree(x_d);
	hipFree(y_d);
	free(x);
	free(y);
	free(y_dup);


	std::cout << "Lazy, you are!\n";
	std::cout << "Write code, you must\n";

	return 0;
}

/* 
 Some helpful definitions

 generateThreadCount is the number of threads spawned initially. Each thread is responsible for sampleSize points. 
 *pSums is a pointer to an array that holds the number of 'hit' points for each thread. The length of this array is pSumSize.

 reduceThreadCount is the number of threads used to reduce the partial sums.
 *totals is a pointer to an array that holds reduced values.
 reduceSize is the number of partial sums that each reduceThreadCount reduces.

*/

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	//	Insert code here
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 0;

	//      Insert code here
	std::cout << "Sneaky, you are ...\n";
	std::cout << "Compute pi, you must!\n";
	return approxPi;
}
